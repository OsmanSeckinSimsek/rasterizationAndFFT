#include "hip/hip_runtime.h"
/*
 * Cornerstone octree
 *
 * Copyright (c) 2024 CSCS, ETH Zurich
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: MIT License
 */

/*! @file
 * @brief  Tests for warp-level primitives
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 * @author Felix Thaler <thaler@cscs.ch>
 */

#include "gtest/gtest.h"

#include <algorithm>
#include <array>
#include <cstddef>
#include <functional>
#include <iomanip>
#include <numeric>
#include <random>
#include <ranges>
#include <ranges>
#include <sstream>
#include <tuple>
#include <type_traits>
#include <utility>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "cstone/cuda/thrust_util.cuh"
#include "cstone/primitives/warpscan.cuh"

using namespace cstone;

__device__ unsigned globalIndex()
{
    const auto blockIndex = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    return blockIndex * blockDim.x * blockDim.y * blockDim.z + threadIdx.x + threadIdx.y * blockDim.x +
           threadIdx.z * blockDim.x * blockDim.y;
}

template<class InputT, class OutputT = InputT, class F>
__global__ void applyWarpCollectiveFunction(InputT* const input, OutputT* output, F f)
{
    const unsigned index = globalIndex();
    output[index]        = f(input[index]);
}

struct SomeStruct
{
    int a;
    float b;
    double c;
    bool d;

    bool operator==(SomeStruct const& other) const
    {
        return a == other.a && b == other.b && c == other.c && d == other.d;
    }
};

std::ostream& operator<<(std::ostream& out, SomeStruct const& s)
{
    out << "SomeStruct {" << s.a << ", " << s.b << ", " << s.c << ", " << s.d << "}";
    return out;
}

template<class T>
std::tuple<dim3, dim3, thrust::host_vector<T>> warpCollectiveFunctionTestData()
{
    // Note: we use 3D thread blocks here to test proper lane indexing in multi-D blocks (test data is still 1D)
    const dim3 numBlocks = {5, 2, 3};
    const dim3 blockSize = {GpuConfig::warpSize / 4, 2, 6};

    thrust::host_vector<T> data(blockSize.x * blockSize.y * blockSize.z * numBlocks.x * numBlocks.y * numBlocks.z, T{});

    std::default_random_engine eng;
    if constexpr (std::is_same_v<T, SomeStruct>)
    {
        using IntDist    = std::uniform_int_distribution<int>;
        using FloatDist  = std::uniform_real_distribution<float>;
        using DoubleDist = std::uniform_real_distribution<double>;
        using BoolDist   = std::bernoulli_distribution;

        auto randomInt    = std::bind(IntDist{}, std::ref(eng));
        auto randomFloat  = std::bind(FloatDist{}, std::ref(eng));
        auto randomDouble = std::bind(DoubleDist{}, std::ref(eng));
        auto randomBool   = std::bind(BoolDist{}, std::ref(eng));

        std::generate(data.begin(), data.end() - GpuConfig::warpSize,
                      [&] { return SomeStruct{randomInt(), randomFloat(), randomDouble(), randomBool()}; });
    }
    else
    {
        using Dist = std::conditional_t<
            std::is_floating_point_v<T>, std::uniform_real_distribution<T>,
            std::conditional_t<std::is_same_v<T, bool>, std::bernoulli_distribution, std::uniform_int_distribution<T>>>;

        std::generate(data.begin(), data.end() - GpuConfig::warpSize, std::bind(Dist{}, std::ref(eng)));
    }

    return {std::move(numBlocks), std::move(blockSize), std::move(data)};
}

template<class T>
using WarpSpan = std::span<T, GpuConfig::warpSize>;

template<class InputT, class OutputT, class WarpF>
void verifyWarpCollectiveFunctionOutput(thrust::host_vector<InputT> const& input,
                                        WarpF warpF,
                                        thrust::host_vector<OutputT> const& output)
{
    ASSERT_EQ(input.size(), output.size());
    ASSERT_EQ(input.size() % GpuConfig::warpSize, 0);
    for (std::size_t i = 0; i < input.size(); i += GpuConfig::warpSize)
    {
        WarpSpan<const InputT> warpInput(&input[i], &input[i + GpuConfig::warpSize]);
        WarpSpan<const OutputT> warpOutput(&output[i], &output[i + GpuConfig::warpSize]);
        std::array<OutputT, GpuConfig::warpSize> expectedWarpOutput;

        warpF(warpInput, WarpSpan<OutputT>(expectedWarpOutput));

        if (!std::ranges::equal(warpOutput, expectedWarpOutput))
        {
            std::ostringstream failures;
            for (unsigned i = 0; i < GpuConfig::warpSize; ++i)
                failures << "Lane " << std::setw(2) << i << " - input: " << warpInput[i]
                         << ", output: " << warpOutput[i] << ", expected output: " << expectedWarpOutput[i] << "\n";

            ADD_FAILURE() << failures.view();
        }
    }
}

/* Helper to test warp-collective functions on the GPU. InputT/OutputT are per-thread input/output types
 * The functor f will be invoked on device and must also provide a reference implementation for a single warp on the
 * host to verify against. I.e., f must:
 * - be a device-callable functor, taking a single argument,
 * - have static member F::reference which is a functor with signature
 *   (WarpSpan<const InputT>, WarpSpan<OutputT>) -> void.
 */
template<class InputT, class OutputT = InputT, class F>
void testWarpCollectiveFunction(F f)
{
    const auto [numBlocks, blockSize, input] = warpCollectiveFunctionTestData<InputT>();

    thrust::device_vector<InputT> deviceInput = input;
    thrust::device_vector<OutputT> deviceOutput(input.size());
    applyWarpCollectiveFunction<<<numBlocks, blockSize>>>(rawPtr(deviceInput), rawPtr(deviceOutput), f);
    checkGpuErrors(hipDeviceSynchronize());

    thrust::host_vector<OutputT> output = deviceOutput;
    verifyWarpCollectiveFunctionOutput(input, F::reference, output);
}

struct WarpLaneIndex
{
    __device__ unsigned operator()(unsigned /* unused */) const { return laneIndex(); }

    static constexpr auto reference = [](WarpSpan<const unsigned> /* unused */, WarpSpan<unsigned> output)
    { std::iota(output.begin(), output.end(), 0u); };
};

TEST(WarpScan, laneIndex) { testWarpCollectiveFunction<unsigned>(WarpLaneIndex{}); }

template<int Src>
struct WarpShflSync
{
    template<class T>
    __device__ T operator()(T x) const
    {
        return shflSync(x, Src);
    };

    static constexpr auto reference = []<class T>(WarpSpan<const T> input, WarpSpan<T> output)
    { std::ranges::fill(output, input[Src]); };
};

TEST(WarpScan, shflSync)
{
    testWarpCollectiveFunction<int>(WarpShflSync<GpuConfig::warpSize / 10>{});
    testWarpCollectiveFunction<int>(WarpShflSync<GpuConfig::warpSize - 1>{});
    testWarpCollectiveFunction<float>(WarpShflSync<GpuConfig::warpSize / 3>{});
    testWarpCollectiveFunction<float>(WarpShflSync<GpuConfig::warpSize - 1>{});
    testWarpCollectiveFunction<double>(WarpShflSync<GpuConfig::warpSize / 7>{});
    testWarpCollectiveFunction<double>(WarpShflSync<GpuConfig::warpSize - 1>{});
    testWarpCollectiveFunction<SomeStruct>(WarpShflSync<GpuConfig::warpSize / 2>{});
    testWarpCollectiveFunction<SomeStruct>(WarpShflSync<GpuConfig::warpSize - 1>{});
}

template<GpuConfig::ThreadMask LaneMask>
struct WarpShflXorSync
{
    template<class T>
    __device__ T operator()(T x) const
    {
        return shflXorSync(x, LaneMask);
    };

    static constexpr auto reference = []<class T>(WarpSpan<const T> input, WarpSpan<T> output)
    {
        for (std::size_t i = 0; i < output.size(); ++i)
            output[i] = input[i ^ LaneMask];
    };
};

TEST(WarpScan, shflXorSync)
{
    testWarpCollectiveFunction<int>(WarpShflXorSync<2>{});
    testWarpCollectiveFunction<int>(WarpShflXorSync<4>{});
    testWarpCollectiveFunction<float>(WarpShflXorSync<8>{});
    testWarpCollectiveFunction<float>(WarpShflXorSync<16>{});
    testWarpCollectiveFunction<double>(WarpShflXorSync<2>{});
    testWarpCollectiveFunction<double>(WarpShflXorSync<4>{});
    testWarpCollectiveFunction<SomeStruct>(WarpShflXorSync<8>{});
    testWarpCollectiveFunction<SomeStruct>(WarpShflXorSync<16>{});
}

template<unsigned Delta>
struct WarpShflUpSync
{
    template<class T>
    __device__ T operator()(T x) const
    {
        return shflUpSync(x, Delta);
    };

    static constexpr auto reference = []<class T>(WarpSpan<const T> input, WarpSpan<T> output)
    {
        std::copy_n(input.begin(), Delta, output.begin());
        std::copy_n(input.begin(), GpuConfig::warpSize - Delta, output.begin() + Delta);
    };
};

TEST(WarpScan, shflUpSync)
{
    testWarpCollectiveFunction<int>(WarpShflUpSync<1>{});
    testWarpCollectiveFunction<int>(WarpShflUpSync<2>{});
    testWarpCollectiveFunction<float>(WarpShflUpSync<3>{});
    testWarpCollectiveFunction<float>(WarpShflUpSync<4>{});
    testWarpCollectiveFunction<double>(WarpShflUpSync<5>{});
    testWarpCollectiveFunction<double>(WarpShflUpSync<6>{});
    testWarpCollectiveFunction<SomeStruct>(WarpShflUpSync<7>{});
    testWarpCollectiveFunction<SomeStruct>(WarpShflUpSync<8>{});
}

template<unsigned Delta>
struct WarpShflDownSync
{
    template<class T>
    __device__ T operator()(T x) const
    {
        return shflDownSync(x, Delta);
    };

    static constexpr auto reference = []<class T>(WarpSpan<const T> input, WarpSpan<T> output)
    {
        std::copy_n(input.begin() + Delta, GpuConfig::warpSize - Delta, output.begin());
        std::copy_n(input.end() - Delta, Delta, output.end() - Delta);
    };
};

TEST(WarpScan, shflDownSync)
{
    testWarpCollectiveFunction<int>(WarpShflDownSync<1>{});
    testWarpCollectiveFunction<int>(WarpShflDownSync<2>{});
    testWarpCollectiveFunction<float>(WarpShflDownSync<3>{});
    testWarpCollectiveFunction<float>(WarpShflDownSync<4>{});
    testWarpCollectiveFunction<double>(WarpShflDownSync<5>{});
    testWarpCollectiveFunction<double>(WarpShflDownSync<6>{});
    testWarpCollectiveFunction<SomeStruct>(WarpShflDownSync<7>{});
    testWarpCollectiveFunction<SomeStruct>(WarpShflDownSync<8>{});
}

struct WarpBallotSync
{
    __device__ GpuConfig::ThreadMask operator()(bool x) const { return ballotSync(x); };

    static constexpr auto reference = [](WarpSpan<const bool> input, WarpSpan<GpuConfig::ThreadMask> output)
    {
        GpuConfig::ThreadMask result = 0;
        for (std::size_t i = 0; i < output.size(); ++i)
            result |= GpuConfig::ThreadMask(input[i]) << i;
        std::ranges::fill(output, result);
    };
};

TEST(WarpScan, ballotSync) { testWarpCollectiveFunction<bool, GpuConfig::ThreadMask>(WarpBallotSync{}); }

struct WarpAnySync
{
    __device__ bool operator()(bool x) const { return anySync(x); };

    static constexpr auto reference = [](WarpSpan<const bool> input, WarpSpan<bool> output)
    { std::ranges::fill(output, std::accumulate(input.begin(), input.end(), false, std::logical_or<bool>{})); };
};

TEST(WarpScan, anySync) { testWarpCollectiveFunction<bool>(WarpAnySync{}); }

struct WarpMin
{
    template<class T>
    __device__ T operator()(T x) const
    {
        return warpMin(x);
    }

    static constexpr auto reference = []<class T>(WarpSpan<const T> input, WarpSpan<T> output)
    { std::ranges::fill(output, *std::ranges::min_element(input)); };
};

TEST(WarpScan, warpMin)
{
    testWarpCollectiveFunction<int>(WarpMin{});
    testWarpCollectiveFunction<float>(WarpMin{});
    testWarpCollectiveFunction<double>(WarpMin{});
}

struct WarpMax
{
    template<class T>
    __device__ T operator()(T x) const
    {
        return warpMax(x);
    }

    static constexpr auto reference = []<class T>(WarpSpan<const T> input, WarpSpan<T> output)
    { std::ranges::fill(output, *std::ranges::max_element(input)); };
};

TEST(WarpScan, warpMax)
{
    testWarpCollectiveFunction<int>(WarpMax{});
    testWarpCollectiveFunction<float>(WarpMax{});
    testWarpCollectiveFunction<double>(WarpMax{});
}

struct WarpBitwiseOr
{
    template<class T>
    __device__ T operator()(T x) const
    {
        return warpBitwiseOr(x);
    }

    static constexpr auto reference = []<class T>(WarpSpan<const T> input, WarpSpan<T> output)
    { std::ranges::fill(output, std::accumulate(input.begin(), input.end(), T(0), std::bit_or<T>{})); };
};

TEST(WarpScan, warpBitwiseOr)
{
    testWarpCollectiveFunction<int>(WarpBitwiseOr{});
    testWarpCollectiveFunction<unsigned>(WarpBitwiseOr{});
}

struct WarpInclusiveScanInt
{
    __device__ int operator()(int x) const { return inclusiveScanInt(x); }

    static constexpr auto reference = [](WarpSpan<const int> input, WarpSpan<int> output)
    { std::inclusive_scan(input.begin(), input.end(), output.begin()); };
};

TEST(WarpScan, inclusiveScanInt) { testWarpCollectiveFunction<int>(WarpInclusiveScanInt{}); }

struct WarpExclusiveScanBool
{
    __device__ int operator()(bool x) const { return exclusiveScanBool(x); }

    static constexpr auto reference = [](WarpSpan<const bool> input, WarpSpan<int> output)
    { std::exclusive_scan(input.begin(), input.end(), output.begin(), 0, std::plus<int>()); };
};

TEST(WarpScan, exclusiveScanBool) { testWarpCollectiveFunction<bool, int>(WarpExclusiveScanBool{}); }

struct WarpReduceBool
{
    __device__ int operator()(bool x) const { return reduceBool(x); }

    static constexpr auto reference = [](WarpSpan<const bool> input, WarpSpan<int> output)
    { std::ranges::fill(output, std::accumulate(input.begin(), input.end(), 0)); };
};

TEST(WarpScan, reduceBool) { testWarpCollectiveFunction<bool, int>(WarpReduceBool{}); }

template<int Carry>
struct WarpInclusiveSegscanInt
{
    __device__ int operator()(int x) const { return inclusiveSegscanInt(x, Carry); }

    static constexpr auto reference = [](WarpSpan<const int> input, WarpSpan<int> output)
    {
        int result = Carry;
        for (std::size_t i = 0; i < input.size(); ++i)
        {
            result    = input[i] < 0 ? -input[i] - 1 : result + input[i];
            output[i] = result;
        }
    };
};

TEST(WarpScan, inclusiveSegscanInt)
{
    testWarpCollectiveFunction<int>(WarpInclusiveSegscanInt<1>{});
    testWarpCollectiveFunction<int>(WarpInclusiveSegscanInt<42>{});
    testWarpCollectiveFunction<int>(WarpInclusiveSegscanInt<-42>{});
}

struct WarpStreamCompact
{
    template<class T>
    __device__ T operator()(T x) const
    {
        __shared__ T buffer[GpuConfig::warpSize * 3];
        T* tmp            = buffer + GpuConfig::warpSize * (threadIdx.z / 2);
        const int numKeep = streamCompact(&x, x <= 0, tmp);
        return laneIndex() < numKeep ? x : T(42);
    }

    static constexpr auto reference = []<class T>(WarpSpan<const T> input, WarpSpan<T> output)
    {
        auto [_, out] = std::ranges::copy_if(input, output.begin(), [](auto x) { return x <= 0; });
        std::fill(out, output.end(), 42);
    };
};

TEST(WarpScan, streamCompact)
{
    testWarpCollectiveFunction<int>(WarpStreamCompact{});
    testWarpCollectiveFunction<float>(WarpStreamCompact{});
    testWarpCollectiveFunction<double>(WarpStreamCompact{});
}

struct WarpSpreadSeg8
{
    __device__ int operator()(int x) const { return spreadSeg8(x); }

    static constexpr auto reference = [](WarpSpan<const int> input, WarpSpan<int> output)
    {
        for (std::size_t i = 0; i < output.size(); ++i)
            output[i] = i % 8 == 0 ? input[i / 8] : output[i - 1] + 1;
    };
};

TEST(WarpScan, warpSpreadSeg8) { testWarpCollectiveFunction<int>(WarpSpreadSeg8{}); }

__global__ void applyAtomicMinFloat(float* addr, float value)
{
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    atomicMinFloat(addr, index == 137 ? value : 2025.0f);
}

TEST(WarpScan, atomicMinFloat)
{
    thrust::device_vector<float> d_value(1);

    // check especially corner cases -0.0f, 0.0f
    for (float firstSign : {-1.0f, -0.0f, 0.0f, 1.0f})
        for (float secondSign : {-1.0f, -0.0f, 0.0f, 1.0f})
        {
            d_value[0] = 42.0f * firstSign;
            applyAtomicMinFloat<<<2, 128>>>(rawPtr(d_value), 37.5f * secondSign);
            EXPECT_EQ(float(d_value[0]), std::min(42.0f * firstSign, 37.5f * secondSign));
        }
}

__global__ void applyAtomicMaxFloat(float* addr, float value)
{
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    atomicMaxFloat(addr, index == 137 ? value : -2025.0f);
}

TEST(WarpScan, atomicMaxFloat)
{
    thrust::device_vector<float> d_value(1);

    // check especially corner cases -0.0f, 0.0f
    for (float firstSign : {1.0f, -0.0f, 0.0f, 1.0f})
        for (float secondSign : {-1.0f, -0.0f, 0.0f, 1.0f})
        {
            d_value[0] = 42.0f * firstSign;
            applyAtomicMaxFloat<<<2, 128>>>(rawPtr(d_value), 37.5f * secondSign);
            EXPECT_EQ(float(d_value[0]), std::max(42.0f * firstSign, 37.5f * secondSign));
        }
}
