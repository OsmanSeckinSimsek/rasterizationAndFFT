#include "hip/hip_runtime.h"
/*
 * Cornerstone octree
 *
 * Copyright (c) 2024 CSCS, ETH Zurich
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: MIT License
 */

/*! @file
 * @brief  GPU driver for halo discovery using traversal of an octree
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/primitives/math.hpp"
#include "cstone/traversal/collisions_gpu.h"
#include "cstone/traversal/macs.hpp"

namespace cstone
{

template<class KeyType, class T>
__global__ void findHalosKernel(const KeyType* nodePrefixes,
                                const TreeNodeIndex* childOffsets,
                                const TreeNodeIndex* parents,
                                const Vec3<T>* nodeCenters,
                                const Vec3<T>* nodeSizes,
                                const KeyType* leaves,
                                const Vec3<T>* searchCenters,
                                const Vec3<T>* searchSizes,
                                const Box<T> box,
                                TreeNodeIndex firstNode,
                                TreeNodeIndex lastNode,
                                uint8_t* collisionFlags)
{
    unsigned leafIdx = blockIdx.x * blockDim.x + threadIdx.x + firstNode;

    if (leafIdx < lastNode)
    {
        Vec3<T> tC         = searchCenters[leafIdx];
        Vec3<T> tS         = searchSizes[leafIdx];
        KeyType lowestKey  = leaves[firstNode];
        KeyType highestKey = leaves[lastNode];

        // if the halo box is fully inside the assigned SFC range, we skip collision detection
        if (containedIn(lowestKey, highestKey, tC, tS, box)) { return; }

        // mark all colliding node indices outside [lowestKey:highestKey]
        findCollisions(nodePrefixes, childOffsets, parents, nodeCenters, nodeSizes, tC, tS, box, lowestKey, highestKey,
                       collisionFlags);
    }
}

template<class KeyType, class T>
void findHalosGpu(const KeyType* prefixes,
                  const TreeNodeIndex* childOffsets,
                  const TreeNodeIndex* parents,
                  const Vec3<T>* nodeCenters,
                  const Vec3<T>* nodeSizes,
                  const KeyType* leaves,
                  const Vec3<T>* searchCenters,
                  const Vec3<T>* searchSizes,
                  const Box<T>& box,
                  TreeNodeIndex firstNode,
                  TreeNodeIndex lastNode,
                  uint8_t* collisionFlags)
{
    constexpr unsigned numThreads = 128;
    unsigned numBlocks            = iceil(lastNode - firstNode, numThreads);

    if (numBlocks == 0) { return; }
    findHalosKernel<<<numBlocks, numThreads>>>(prefixes, childOffsets, parents, nodeCenters, nodeSizes, leaves,
                                               searchCenters, searchSizes, box, firstNode, lastNode, collisionFlags);
}

#define FIND_HALOS_GPU(KeyType, T)                                                                                     \
    template void findHalosGpu(const KeyType* prefixes, const TreeNodeIndex* childOffsets,                             \
                               const TreeNodeIndex* parents, const Vec3<T>* nodeCenters, const Vec3<T>* nodeSizes,     \
                               const KeyType* leaves, const Vec3<T>* searchCenters, const Vec3<T>* searchSizes,        \
                               const Box<T>& box, TreeNodeIndex firstNode, TreeNodeIndex lastNode,                     \
                               uint8_t* collisionFlags)

FIND_HALOS_GPU(uint32_t, float);
FIND_HALOS_GPU(uint64_t, float);
FIND_HALOS_GPU(uint64_t, double);

template<class T, class KeyType>
__global__ void markMacsGpuKernel(const KeyType* prefixes,
                                  const TreeNodeIndex* childOffsets,
                                  const TreeNodeIndex* parents,
                                  const Vec4<T>* centers,
                                  const Box<T> box,
                                  const KeyType* focusNodes,
                                  TreeNodeIndex numFocusNodes,
                                  bool limitSource,
                                  uint8_t* markings)
{
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= numFocusNodes) { return; }

    KeyType focusStart = focusNodes[0];
    KeyType focusEnd   = focusNodes[numFocusNodes];

    IBox target    = sfcIBox(sfcKey(focusNodes[tid]), sfcKey(focusNodes[tid + 1]));
    IBox targetExt = IBox(target.xmin() - 1, target.xmax() + 1, target.ymin() - 1, target.ymax() + 1, target.zmin() - 1,
                          target.zmax() + 1);
    if (containedIn(focusStart, focusEnd, targetExt)) { return; }

    auto [targetCenter, targetSize] = centerAndSize<KeyType>(target, box);
    unsigned maxLevel               = maxTreeLevel<KeyType>{};
    if (limitSource) { maxLevel = stl::max(int(treeLevel(focusNodes[tid + 1] - focusNodes[tid])) - 1, 0); }
    markMacPerBox(targetCenter, targetSize, maxLevel, prefixes, childOffsets, parents, centers, box, focusStart,
                  focusEnd, markings);
}

template<class T, class KeyType>
void markMacsGpu(const KeyType* prefixes,
                 const TreeNodeIndex* childOffsets,
                 const TreeNodeIndex* parents,
                 const Vec4<T>* centers,
                 const Box<T>& box,
                 const KeyType* focusNodes,
                 TreeNodeIndex numFocusNodes,
                 bool limitSource,
                 uint8_t* markings)
{
    constexpr unsigned numThreads = 128;
    unsigned numBlocks            = iceil(numFocusNodes, numThreads);

    if (numFocusNodes)
    {
        markMacsGpuKernel<<<numBlocks, numThreads>>>(prefixes, childOffsets, parents, centers, box, focusNodes,
                                                     numFocusNodes, limitSource, markings);
    }
}

#define MARK_MACS_GPU(KeyType, T)                                                                                      \
    template void markMacsGpu(const KeyType* prefixes, const TreeNodeIndex* childOffsets,                              \
                              const TreeNodeIndex* parents, const Vec4<T>* centers, const Box<T>& box,                 \
                              const KeyType* focusNodes, TreeNodeIndex numFocusNodes, bool limitSource,                \
                              uint8_t* markings)

MARK_MACS_GPU(uint64_t, double);
MARK_MACS_GPU(uint64_t, float);
MARK_MACS_GPU(unsigned, float);

} // namespace cstone
