#include "hip/hip_runtime.h"
/*
 * Cornerstone octree
 *
 * Copyright (c) 2024 CSCS, ETH Zurich
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: MIT License
 */

/*! @file
 * @brief  Compute the internal part of a cornerstone octree on the GPU
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 *
 */

#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include "cstone/primitives/math.hpp"
#include "cstone/primitives/primitives_gpu.h"
#include "cstone/sfc/common.hpp"
#include "cstone/tree/octree_gpu.h"

namespace cstone
{

/*! @brief combine internal and leaf tree parts into a single array with the nodeKey prefixes
 *
 * @tparam     KeyType           unsigned 32- or 64-bit integer
 * @param[in]  leaves            cornerstone SFC keys, length numLeafNodes + 1
 * @param[in]  numInternalNodes  number of internal octree nodes
 * @param[in]  numLeafNodes      total number of nodes
 * @param[in]  binaryToOct       translation map from binary to octree nodes
 * @param[out] prefixes          output octree SFC keys, length @p numInternalNodes + numLeafNodes
 *                               NOTE: keys are prefixed with Warren-Salmon placeholder bits!
 * @param[out] nodeOrder         iota 0,1,2,3,... sequence for later use, length same as @p prefixes
 */
template<class KeyType>
__global__ void createUnsortedLayout(const KeyType* leaves,
                                     TreeNodeIndex numInternalNodes,
                                     TreeNodeIndex numLeafNodes,
                                     KeyType* prefixes,
                                     TreeNodeIndex* nodeOrder)
{
    int tid = int(blockDim.x * blockIdx.x + threadIdx.x);
    if (tid < numLeafNodes)
    {
        KeyType key                       = leaves[tid];
        unsigned level                    = treeLevel(leaves[tid + 1] - key);
        prefixes[tid + numInternalNodes]  = encodePlaceholderBit(key, 3 * level);
        nodeOrder[tid + numInternalNodes] = tid + numInternalNodes;

        unsigned prefixLength = commonPrefix(key, leaves[tid + 1]);
        if (prefixLength % 3 == 0 && tid < numLeafNodes - 1)
        {
            TreeNodeIndex octIndex = (tid + binaryKeyWeight(key, prefixLength / 3)) / 7;
            prefixes[octIndex]     = encodePlaceholderBit(key, prefixLength);
            nodeOrder[octIndex]    = octIndex;
        }
    }
}

/*! @brief extract parent/child relationships from binary tree and translate to sorted order
 *
 * @tparam     KeyType           unsigned 32- or 64-bit integer
 * @param[in]  prefixes          octree node prefixes in Warren-Salmon format
 * @param[in]  numInternalNodes  number of internal octree nodes
 * @param[in]  leafToInternal    translation map from unsorted layout to level/SFC sorted octree layout
 *                               length is total number of octree nodes, internal + leaves
 * @param[in]  levelRange        indices of the first node at each level
 * @param[out] childOffsets      octree node index of first child for each node, length is total number of nodes
 * @param[out] parents           parent index of for each node which is the first of 8 siblings
 *                               i.e. the parent of node i is stored at parents[(i - 1)/8]
 */
template<class KeyType>
__global__ void linkTree(const KeyType* prefixes,
                         TreeNodeIndex numInternalNodes,
                         const TreeNodeIndex* leafToInternal,
                         const TreeNodeIndex* levelRange,
                         TreeNodeIndex* childOffsets,
                         TreeNodeIndex* parents)
{
    // loop over octree nodes index in unsorted layout A
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numInternalNodes)
    {
        TreeNodeIndex idxA    = leafToInternal[tid];
        KeyType prefix        = prefixes[idxA];
        KeyType nodeKey       = decodePlaceholderBit(prefix);
        unsigned prefixLength = decodePrefixLength(prefix);
        unsigned level        = prefixLength / 3;
        assert(level < maxTreeLevel<KeyType>{});

        KeyType childPrefix = encodePlaceholderBit(nodeKey, prefixLength + 3);

        TreeNodeIndex leafSearchStart = levelRange[level + 1];
        TreeNodeIndex leafSearchEnd   = levelRange[level + 2];
        TreeNodeIndex childIdx =
            stl::lower_bound(prefixes + leafSearchStart, prefixes + leafSearchEnd, childPrefix) - prefixes;

        if (childIdx != leafSearchEnd && childPrefix == prefixes[childIdx])
        {
            childOffsets[idxA] = childIdx;
            // We only store the parent once for every group of 8 siblings.
            // This works as long as each node always has 8 siblings.
            // Subtract one because the root has no siblings.
            parents[(childIdx - 1) / 8] = idxA;
        }
    }
}

//! @brief determine the octree subdivision level boundaries
template<class KeyType>
__global__ void getLevelRange(const KeyType* nodeKeys, TreeNodeIndex numNodes, TreeNodeIndex* levelRange)
{
    unsigned level    = blockIdx.x;
    auto it           = stl::lower_bound(nodeKeys, nodeKeys + numNodes, encodePlaceholderBit(KeyType(0), 3 * level));
    levelRange[level] = TreeNodeIndex(it - nodeKeys);

    if (level == maxTreeLevel<KeyType>{} + 1) { levelRange[level] = numNodes; }
}

//! @brief computes the inverse of the permutation given by @p order and then subtract @p numInternalNodes from it
__global__ void
invertOrder(TreeNodeIndex* order, TreeNodeIndex* inverseOrder, TreeNodeIndex numNodes, TreeNodeIndex numInternalNodes)
{
    int tid = int(blockDim.x * blockIdx.x + threadIdx.x);
    if (tid < numNodes)
    {
        inverseOrder[order[tid]] = tid;
        order[tid] -= numInternalNodes;
    }
}

template<class KeyType>
void buildOctreeGpu(const KeyType* cstoneTree,
                    OctreeView<KeyType> d,
                    std::span<KeyType> keyBuf,
                    std::span<TreeNodeIndex> valueBuf,
                    std::span<char> cubTmp)
{
    constexpr unsigned numThreads = 256;

    TreeNodeIndex numNodes = d.numInternalNodes + d.numLeafNodes;
    createUnsortedLayout<<<iceil(numNodes, numThreads), numThreads>>>(cstoneTree, d.numInternalNodes, d.numLeafNodes,
                                                                      d.prefixes, d.internalToLeaf);

    assert(keyBuf.size() == d.numNodes && valueBuf.size() == d.numNodes);
    sortByKeyGpu(d.prefixes, d.prefixes + numNodes, d.internalToLeaf, keyBuf.data(), valueBuf.data(), cubTmp.data(),
                 cubTmp.size());

    invertOrder<<<iceil(numNodes, numThreads), numThreads>>>(d.internalToLeaf, d.leafToInternal, numNodes,
                                                             d.numInternalNodes);
    getLevelRange<<<maxTreeLevel<KeyType>{} + 2, 1>>>(d.prefixes, numNodes, d.d_levelRange);
    memcpyD2H(d.d_levelRange, maxTreeLevel<KeyType>{} + 2, d.levelRange);

    thrust::fill(thrust::device, d.childOffsets, d.childOffsets + numNodes, 0);
    if (d.numInternalNodes)
    {
        linkTree<<<iceil(d.numInternalNodes, numThreads), numThreads>>>(
            d.prefixes, d.numInternalNodes, d.leafToInternal, d.d_levelRange, d.childOffsets, d.parents);
    }
}

template void
buildOctreeGpu(const uint32_t*, OctreeView<uint32_t>, std::span<uint32_t>, std::span<TreeNodeIndex>, std::span<char>);
template void
buildOctreeGpu(const uint64_t*, OctreeView<uint64_t>, std::span<uint64_t>, std::span<TreeNodeIndex>, std::span<char>);

template<class KeyType>
void buildOctreeGpu(const KeyType* cstoneTree, OctreeView<KeyType> d)
{
    KeyType* keyBuf;
    TreeNodeIndex* valueBuf;
    char* cubTmp;
    uint64_t tmpStorage = sortByKeyTempStorage<KeyType, TreeNodeIndex>(d.numNodes);
    checkGpuErrors(hipMalloc(&keyBuf, sizeof(KeyType) * d.numNodes));
    checkGpuErrors(hipMalloc(&valueBuf, sizeof(TreeNodeIndex) * d.numNodes));
    checkGpuErrors(hipMalloc(&cubTmp, tmpStorage));

    buildOctreeGpu(cstoneTree, d, {keyBuf, size_t(d.numNodes)}, {valueBuf, size_t(d.numNodes)}, {cubTmp, tmpStorage});

    checkGpuErrors(hipFree(keyBuf));
    checkGpuErrors(hipFree(valueBuf));
    checkGpuErrors(hipFree(cubTmp));
}

template void buildOctreeGpu(const uint32_t*, OctreeView<uint32_t>);
template void buildOctreeGpu(const uint64_t*, OctreeView<uint64_t>);

__global__ void upsweepSumKernel(TreeNodeIndex firstCell,
                                 TreeNodeIndex lastCell,
                                 const TreeNodeIndex* childOffsets,
                                 LocalIndex* nodeCounts)
{
    const int cellIdx = blockIdx.x * blockDim.x + threadIdx.x + firstCell;
    if (cellIdx >= lastCell) return;

    TreeNodeIndex firstChild = childOffsets[cellIdx];

    if (firstChild) { nodeCounts[cellIdx] = NodeCount<LocalIndex>{}(cellIdx, firstChild, nodeCounts); }
}

void upsweepSumGpu(int numLevels,
                   const TreeNodeIndex* levelRange,
                   const TreeNodeIndex* childOffsets,
                   LocalIndex* nodeCounts)
{
    constexpr int numThreads = 128;

    for (int level = numLevels - 1; level >= 0; level--)
    {
        int numCellsLevel = levelRange[level + 1] - levelRange[level];
        int numBlocks     = (numCellsLevel - 1) / numThreads + 1;
        if (numCellsLevel)
        {
            upsweepSumKernel<<<numBlocks, numThreads>>>(levelRange[level], levelRange[level + 1], childOffsets,
                                                        nodeCounts);
        }
    }
}

template<class KeyType>
__global__ void locateNodesKernel(const KeyType* k1,
                                  const KeyType* k2,
                                  const KeyType* nodeKeys,
                                  const TreeNodeIndex* lvlRange,
                                  TreeNodeIndex* indices)
{
    LocalIndex tid  = blockIdx.x * blockDim.x + threadIdx.x;
    TreeNodeIndex n = k2 - k1 - 1;
    if (tid < n) { indices[tid] = locateNode(k1[tid], k1[tid + 1], nodeKeys, lvlRange); }
}

template<class KeyType>
void locateNodesGpu(const KeyType* k1,
                    const KeyType* k2,
                    const KeyType* nodeKeys,
                    const TreeNodeIndex* lvlRange,
                    TreeNodeIndex* indices)
{
    int numThreads = 256;
    int numBlocks  = iceil(k2 - k1 - 1, numThreads);
    if (numBlocks == 0) { return; }
    locateNodesKernel<<<numBlocks, numThreads>>>(k1, k2, nodeKeys, lvlRange, indices);
}

template void locateNodesGpu(const uint32_t* k1,
                             const uint32_t* k2,
                             const uint32_t* nodeKeys,
                             const TreeNodeIndex* lvlRange,
                             TreeNodeIndex* indices);
template void locateNodesGpu(const uint64_t* k1,
                             const uint64_t* k2,
                             const uint64_t* nodeKeys,
                             const TreeNodeIndex* lvlRange,
                             TreeNodeIndex* indices);

template<class KeyType>
__global__ void locateNodesKernel(const KeyType* k1,
                                  const TreeNodeIndex* map,
                                  size_t n,
                                  const KeyType* nodeKeys,
                                  const TreeNodeIndex* lvlRange,
                                  TreeNodeIndex* indices)
{
    LocalIndex tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) { indices[tid] = locateNode(k1[map[tid]], nodeKeys, lvlRange); }
}

template<class KeyType>
void locateNodesGpu(const KeyType* k1,
                    const TreeNodeIndex* map,
                    size_t n,
                    const KeyType* nodeKeys,
                    const TreeNodeIndex* lvlRange,
                    TreeNodeIndex* indices)
{
    int numThreads = 256;
    int numBlocks  = iceil(n, numThreads);
    if (numBlocks == 0) { return; }
    locateNodesKernel<<<numBlocks, numThreads>>>(k1, map, n, nodeKeys, lvlRange, indices);
}

template void locateNodesGpu(const uint32_t* k1,
                             const TreeNodeIndex* map,
                             size_t n,
                             const uint32_t* nodeKeys,
                             const TreeNodeIndex* lvlRange,
                             TreeNodeIndex* indices);
template void locateNodesGpu(const uint64_t* k1,
                             const TreeNodeIndex* map,
                             size_t n,
                             const uint64_t* nodeKeys,
                             const TreeNodeIndex* lvlRange,
                             TreeNodeIndex* indices);

} // namespace cstone
